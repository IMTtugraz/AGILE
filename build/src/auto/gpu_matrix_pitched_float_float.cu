// Copyright (C) 2010-2011 Institute of Medical Engineering,
// Graz University of Technology
//
// This program is free software; you can redistribute it and/or modify it under
// the terms of the GNU General Public License as published by the Free Software
// Foundation; either version 3 of the License, or (at your option) any later
// version.
//
// This program is distributed in the hope that it will be useful, but
// WITHOUT ANY WARRANTY; without even the implied warranty of MERCHANTABILITY
// or FITNESS FOR A PARTICULAR PURPOSE. See the GNU General Public License for
// more details.
//
// You should have received a copy of the GNU General Public License along with
// this program; if not, see <http://www.gnu.org/licenses>.

// $Id$

/* This file was generated automatically by CMake. You have to modify '/home2/GIT/AGILE/src/gpu_matrix_pitched.cu.in' if you want to make changes. */

#define TType1IsComplex 0
#define TType2IsComplex 0

#include "agile/gpu_config.hpp"
#include "agile/gpu_type_traits.hpp"
#include <hip/hip_runtime.h>

// Unfortunately, textures have to have file scope, which is why we have to
// use this ugly preprocessor definitions.
#define AGILE_TEXTURE agile_matrix_texture_floatfloat
texture<agile::to_tuple_type<float >::texture_type> AGILE_TEXTURE;

#define AGILE_TEXTURE_2D agile_matrix_texture_2d_floatfloat
texture<agile::to_tuple_type<float >::texture_type, 2> AGILE_TEXTURE_2D;

#include "gpu_matrix_pitched.ipp"

namespace agile
{
  template void multiply<float, float >(
    const GPUMatrixPitched<float >& A, const GPUVector<float >& x,
    GPUVector<typename promote<float, float >::type>& y);

  template void multiply<float, float >(
    const GPUVector<float >& x, const GPUMatrixPitched<float >& A,
    GPUVector<typename promote<float, float >::type>& y);

  template void multiplyElementwise<float, float >(
    const GPUMatrixPitched<float >& A, const GPUMatrixPitched<float >& B,
    GPUMatrixPitched<typename promote<float, float >::type>& Z);

  template void scale<float, float >(
    const float& alpha, const GPUMatrixPitched<float >& A,
    GPUMatrixPitched<typename promote<float, float >::type>& B);

#if !TType2IsComplex

  // second type may not be complex because, complex is already in signature
  template void interp2d<float, float >(
    const GPUMatrixPitched<float >& M,
    const GPUVector<std::complex<float > >& pos,
    GPUVector<float >& res);

#endif  // TType2IsComplex

} // namespace agile

// End of $Id: gpu_matrix.cu.in 376 2010-02-11 13:16:45Z freiberger $.
