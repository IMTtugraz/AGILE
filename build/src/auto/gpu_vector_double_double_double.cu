#include "hip/hip_runtime.h"
// Copyright (C) 2010-2011 Institute of Medical Engineering,
// Graz University of Technology
//
// This program is free software; you can redistribute it and/or modify it under
// the terms of the GNU General Public License as published by the Free Software
// Foundation; either version 3 of the License, or (at your option) any later
// version.
//
// This program is distributed in the hope that it will be useful, but
// WITHOUT ANY WARRANTY; without even the implied warranty of MERCHANTABILITY
// or FITNESS FOR A PARTICULAR PURPOSE. See the GNU General Public License for
// more details.
//
// You should have received a copy of the GNU General Public License along with
// this program; if not, see <http://www.gnu.org/licenses>.

// $Id: gpu_vector.cu.in 452 2011-05-31 12:00:18Z freiberger $

/* This file was generated automatically by CMake. You have to modify '/home2/GIT/AGILE/src/gpu_vector.cu.in' if you want to make changes. */

#define TType2EqualTType3 1
#define TType1IsComplex 0
#define TType2IsComplex 0

#include "agile/gpu_config.hpp"
#include "agile/gpu_type_traits.hpp"
#include <hip/hip_runtime.h>

// Unfortunately, textures have to have file scope, which is why we have to
// use this ugly preprocessor definitions.
#define AGILE_TEXTURE agile_matrix_texture_doubledouble
texture<agile::to_tuple_type<double >::texture_type> AGILE_TEXTURE;

#define AGILE_TEXTURE_2D agile_matrix_texture_2d_doubledouble
texture<agile::to_tuple_type<double >::texture_type, 2> AGILE_TEXTURE_2D;


#include "gpu_vector.ipp"

namespace agile
{
  
  template 
  void copy<double >(const GPUVector<double >& x, GPUVector<double >& y);
  
  template 
  void maxElement<double >(const GPUVector<double >& x, int* maxVal);

namespace lowlevel
{
  // **************************************************************************
  // functions that depend on one type only
  // **************************************************************************


#if TType2EqualTType3
#if !TType2IsComplex


  template void interpolate2d<double, double >(
    const double* src, unsigned numColumns, unsigned numRows,
    bool reshapeRowMajor, const std::complex<double>* pos,
    double* res, unsigned size);


  template void fftshift<double >(double* x, unsigned size1,
                                    unsigned size2);

  template void ifftshift<double >(double* x, unsigned size1,
                                    unsigned size2);


  template void absVector<double >(
      const double* x,
      typename to_real_type<double >::type* y, unsigned size);

  template void meshgrid<double >(
      double* mesh_x, double* mesh_y,
      const double* x, unsigned x_size, const double* y, unsigned y_size);


  template void imag<double >(
    const double* x,
    typename to_real_type<double >::type* y, unsigned size);

  template typename to_real_type<double >::type norm1(
    const double* x, unsigned size);
  
  template typename to_real_type<double >::type norm2(
    const double* x, unsigned size);

  template void real<double >(
    const double* x,
    typename to_real_type<double >::type* y, unsigned size);

  template void setVectorConstant<double >(
    const double& value, double* x, unsigned size);

  template void pattern<double >(
      const double* x, typename to_real_type<double >::type* z, unsigned size);

  template void diff<double >(
    const unsigned dim, const unsigned x_size, const double* x, double* y, unsigned size);
  
  template void difftrans<double >(
    const unsigned dim, const unsigned x_size, const double* x, double* y, unsigned size);

  template void diff3<double >(
    const unsigned dim, const unsigned x_size, const unsigned y_size, const double* x, double* y, unsigned size, bool borderWrap);
  
  template void diff3trans<double >(
    const unsigned dim, const unsigned x_size, const unsigned y_size, const double* x, double* y, unsigned size, bool borderWrap);
  
  template void bdiff3<double >(
    const unsigned dim, const unsigned x_size, const unsigned y_size, const double* x, double* y, unsigned size, bool borderWrap);
  
  template void bdiff3trans<double >(
    const unsigned dim, const unsigned x_size, const unsigned y_size, const double* x, double* y, unsigned size, bool borderWrap);

  template void sqrt<double >(const double* x, double* y, unsigned size);

  template void expand_rowdim<double >(const double* x_data, const double* delta_o, const double* delta_u,
                                        unsigned rows, unsigned cols, unsigned row_o, unsigned row_u,
                                        double* z);

  template void expand_coldim<double >(const double* x_data, const double* delta_o, const double* delta_u,
                                        unsigned rows, unsigned cols, unsigned col_o, unsigned col_u,
                                        double* z);

  template void get_content<double >(const double* x_data, unsigned rows, unsigned cols,
                     unsigned row_offset, unsigned col_offset, double* z, unsigned z_rows, unsigned z_cols);

#if !TType1IsComplex

  template void linspace<double >(double* x, unsigned size,
                                    float a, float b);

  template void pow<double, double>(const double& alpha,
                              const double* x,
                              double* y, unsigned size);

#endif  // !TType1IsComplex
#endif  // !TType2IsComplex
#endif  // TType2EqualTType3


  // **************************************************************************
  // functions that depend on two types
  // **************************************************************************
#if TType2EqualTType3

  template void addVector<double, double >(
    const double* x, const double* y,
    typename promote<double, double >::type* z, unsigned size);

  template void divideVector<double, double >(
    const double& alpha, const double* x,
    typename promote<double, double >::type* y, unsigned size);

  template typename promote<double, double >::type getScalarProduct(
    const double* x, const double* y, unsigned size);

  template void multiplyConjElementwise<double, double >(
    const double* x, const double* y,
    typename promote<double, double >::type* z, unsigned size);

  template void multiplyElementwise<double, double >(
    const double* x, const double* y,
    typename promote<double, double >::type* z, unsigned size);

  template void divideElementwise<double, double >(
    const double* x, const double* y,
    typename promote<double, double >::type* z, unsigned size);

  template void scale<double, double >(
    const double& alpha, const double* x,
    typename promote<double, double >::type* y, unsigned size);
    
  template void subVector<double, double >(
    const double* x, const double* y,
    typename promote<double, double >::type* z, unsigned size);

  template void conjVector<double >(
      const double* x,  double* z, unsigned size);
  
  template void expVector<double >(
      const double* x,  double* z, unsigned size);

  template void max<double, double >(
      const double* x1, const double* x2, typename promote<double, double >::type* y, unsigned size);

  template void max<double, double >(
      const double* x1, const double & x2, typename promote<double, double >::type* y, unsigned size);

#if TType1IsComplex
#if !TType2IsComplex
template void phaseVector<double, double >(
    const double* x,
    double* y, unsigned size);
#endif  // TType1IsComplex
#endif  // !TType2IsComplex


#endif  // TType2EqualTType3


  // **************************************************************************
  // functions that depend on three types
  // **************************************************************************

  template void addScaledVector<double, double, double >(
    const double* x, const double& scale, const double* y,
    typename promote<typename promote<double, double >::type,
                     double >::type* z,
    unsigned size);

  template void subScaledVector<double, double, double >(
    const double* x, const double& scale, const double* y,
    typename promote<typename promote<double, double >::type,
                     double >::type* z,
    unsigned size);

} // namespace lowlevel
} // namespace agile

// End of $Id: gpu_vector.cu.in 452 2011-05-31 12:00:18Z freiberger $.
